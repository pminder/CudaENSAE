#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "md5.h"

__global__ void kernel(char * devMd5sum, char * devMessage, size_t size)
{
    md5(devMd5sum, devMessage, size);
}
 
int main()
{
    md5_init();
    printf("MD5 test suite :\n");

    const char * t = "message digest";
    char mdsum[16], mdoutput[32 + 1];
    char * devMd5sum, *devMessage;

    hipMalloc((void**)&devMd5sum, 16 * sizeof(char));
    hipMalloc((void**)&devMessage, 15 * sizeof(char));
    hipMemcpy(devMessage, t, 15 * sizeof(char), hipMemcpyHostToDevice);

    kernel<<<1,1>>>(devMd5sum, devMessage, 14);

    hipMemcpy(mdsum, devMd5sum, 16 * sizeof(char), hipMemcpyDeviceToHost);
    
    printf("md5(\"%s\") = ", t);

    md5_format(mdoutput, mdsum);
    printf("%s\n", mdoutput);
 
    return 0;
}